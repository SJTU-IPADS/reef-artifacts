#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))


#define NUM_BLOCKS 64
#define NUM_TREHAD_PER_BLOCK 128
#define BLOCKDIM_X 4
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 4

#define __activemask() 0

#define CU_NUM 60

__device__ __forceinline__ bool is_first_thread() {
  return threadIdx.x == 0;
}

__device__ __forceinline__ unsigned int get_cu_id() {
  return blockIdx.x % CU_NUM;
}

__device__ __forceinline__ dim3 get_3d_idx(int idx, dim3 dim) {
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__device__  void multiply_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp){
    __shared__ float buffer[1024 * 32 / 4];
    int blockOffset = task_idx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    temp[arrayOffset] = a[threadOffset] * b[arrayOffset];
}

__device__  void add_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp){
    __shared__ float buffer[1024 * 32 / 4];
    int blockOffset = task_idx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    temp[arrayOffset] = a[threadOffset] + b[arrayOffset];
}

extern "C" __global__ void multiply(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(64,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        multiply_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void add(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(64,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        add_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void preemption_proxy(volatile int* stop, volatile int* host, volatile int* device) {
    while(!(*stop)) {
        *device = *host;
        long long start_clock = clock64();
        while (clock64() < (start_clock+10000)) {}
    }
}
