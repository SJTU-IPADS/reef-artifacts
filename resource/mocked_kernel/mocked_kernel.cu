#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))


#define NUM_BLOCKS 64
#define NUM_TREHAD_PER_BLOCK 128
#define BLOCKDIM_X 4
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 4

#define __activemask() 0

extern "C" __global__ void multiply(float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp) {
    __shared__ float buffer[1024 * 32 / 4];
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    temp[arrayOffset] = a[threadOffset] * b[arrayOffset];
}

extern "C" __global__ void add(float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp) {
    __shared__ float buffer[1024 * 32 / 4];
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    temp[arrayOffset] = a[threadOffset] + b[arrayOffset];
}
