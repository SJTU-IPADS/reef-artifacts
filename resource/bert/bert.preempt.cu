#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define CU_NUM 60

__device__ __forceinline__ bool is_first_thread() {
  return threadIdx.x == 0;
}

__device__ __forceinline__ unsigned int get_cu_id() {
  return blockIdx.x % CU_NUM;
}

__device__ __forceinline__ dim3 get_3d_idx(int idx, dim3 dim) {
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__device__  void fused_nn_softmax_1_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_softmax_maxelem, float* __restrict__ placeholder){
  if (((((int)task_idx.x) * 256) + ((int)threadIdx.x)) < 5760) {
    T_softmax_maxelem[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))] = -3.402823e+38f;
  }
  for (int k = 0; k < 480; ++k) {
    if (((((int)task_idx.x) * 256) + ((int)threadIdx.x)) < 5760) {
      T_softmax_maxelem[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))] = max(T_softmax_maxelem[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))], placeholder[((((((int)task_idx.x) * 122880) + (((int)threadIdx.x) * 480)) + k))]);
    }
  }
}

__device__  void fused_reshape_add_add_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_add[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = ((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480))]) + placeholder2[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))]);
    }
  }
}

__device__  void fused_nn_batch_matmul_4_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute){
  float compute_local[8];
  __shared__ float placeholder_shared[720];
  __shared__ float placeholder_d_shared[300];
  float placeholder_shared_local[2];
  float placeholder_d_shared_local[4];
  for (int i_c_init = 0; i_c_init < 2; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 4; ++j_c_init) {
      compute_local[(((i_c_init * 4) + j_c_init))] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 32; ++k_outer) {
    __syncthreads();
    placeholder_shared[(((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)))] = placeholder[((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 1))] = placeholder[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 1))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 2))] = placeholder[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 2))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 15))] = placeholder[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 480))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 16))] = placeholder[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 481))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 17))] = placeholder[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 482))];
    if (((int)threadIdx.y) < 20) {
      placeholder_d_shared[(((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 3)))] = placeholder1[((((((((int)task_idx.z) * 19200) + (((int)task_idx.x) * 9600)) + (((int)threadIdx.y) * 480)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)))];
    }
    if (((int)threadIdx.y) < 20) {
      placeholder_d_shared[((((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 3)) + 1))] = placeholder1[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.x) * 9600)) + (((int)threadIdx.y) * 480)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 1))];
    }
    if (((int)threadIdx.y) < 20) {
      placeholder_d_shared[((((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 3)) + 2))] = placeholder1[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.x) * 9600)) + (((int)threadIdx.y) * 480)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 2))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 15; ++k_inner) {
      placeholder_shared_local[(0)] = placeholder_shared[(((((int)threadIdx.y) * 30) + k_inner))];
      placeholder_shared_local[(1)] = placeholder_shared[((((((int)threadIdx.y) * 30) + k_inner) + 15))];
      placeholder_d_shared_local[(0)] = placeholder_d_shared[(((((int)threadIdx.x) * 60) + k_inner))];
      placeholder_d_shared_local[(1)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 15))];
      placeholder_d_shared_local[(2)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 30))];
      placeholder_d_shared_local[(3)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 45))];
      compute_local[(0)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(0)], compute_local[(0)]);
      compute_local[(1)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(1)], compute_local[(1)]);
      compute_local[(2)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(2)], compute_local[(2)]);
      compute_local[(3)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(3)], compute_local[(3)]);
      compute_local[(4)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(0)], compute_local[(4)]);
      compute_local[(5)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(1)], compute_local[(5)]);
      compute_local[(6)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(2)], compute_local[(6)]);
      compute_local[(7)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(3)], compute_local[(7)]);
    }
  }
  compute[((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)task_idx.x) * 20)) + (((int)threadIdx.x) * 4)))] = compute_local[(0)];
  compute[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)task_idx.x) * 20)) + (((int)threadIdx.x) * 4)) + 1))] = compute_local[(1)];
  compute[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)task_idx.x) * 20)) + (((int)threadIdx.x) * 4)) + 2))] = compute_local[(2)];
  compute[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)task_idx.x) * 20)) + (((int)threadIdx.x) * 4)) + 3))] = compute_local[(3)];
  compute[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)task_idx.x) * 20)) + (((int)threadIdx.x) * 4)) + 40))] = compute_local[(4)];
  compute[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)task_idx.x) * 20)) + (((int)threadIdx.x) * 4)) + 41))] = compute_local[(5)];
  compute[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)task_idx.x) * 20)) + (((int)threadIdx.x) * 4)) + 42))] = compute_local[(6)];
  compute[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)task_idx.x) * 20)) + (((int)threadIdx.x) * 4)) + 43))] = compute_local[(7)];
}

__device__  void fused_nn_batch_matmul_5_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute){
  float compute_local[15];
  __shared__ float placeholder_shared[800];
  __shared__ float placeholder_d_shared[480];
  float placeholder_shared_local[5];
  float placeholder_d_shared_local[3];
  for (int i_c_init = 0; i_c_init < 5; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 3; ++j_c_init) {
      compute_local[(((i_c_init * 3) + j_c_init))] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 4; ++k_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.y) * 50) + ((int)threadIdx.x)))] = placeholder[((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 10))] = placeholder[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 40))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 20))] = placeholder[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 80))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 30))] = placeholder[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 120))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 40))] = placeholder[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 160))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_d_shared[(((((int)threadIdx.y) * 30) + ((int)threadIdx.x)))] = placeholder1[((((((((int)task_idx.z) * 19200) + (((int)task_idx.x) * 1920)) + (((int)threadIdx.y) * 120)) + (k_outer * 10)) + ((int)threadIdx.x)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_d_shared[((((((int)threadIdx.y) * 30) + ((int)threadIdx.x)) + 10))] = placeholder1[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.x) * 1920)) + (((int)threadIdx.y) * 120)) + (k_outer * 10)) + ((int)threadIdx.x)) + 40))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_d_shared[((((((int)threadIdx.y) * 30) + ((int)threadIdx.x)) + 20))] = placeholder1[(((((((((int)task_idx.z) * 19200) + (((int)task_idx.x) * 1920)) + (((int)threadIdx.y) * 120)) + (k_outer * 10)) + ((int)threadIdx.x)) + 80))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 10; ++k_inner) {
      placeholder_shared_local[(0)] = placeholder_shared[(((((int)threadIdx.y) * 50) + k_inner))];
      placeholder_shared_local[(1)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 10))];
      placeholder_shared_local[(2)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 20))];
      placeholder_shared_local[(3)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 30))];
      placeholder_shared_local[(4)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 40))];
      placeholder_d_shared_local[(0)] = placeholder_d_shared[(((((int)threadIdx.x) * 30) + k_inner))];
      placeholder_d_shared_local[(1)] = placeholder_d_shared[((((((int)threadIdx.x) * 30) + k_inner) + 10))];
      placeholder_d_shared_local[(2)] = placeholder_d_shared[((((((int)threadIdx.x) * 30) + k_inner) + 20))];
      compute_local[(0)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(0)], compute_local[(0)]);
      compute_local[(1)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(1)], compute_local[(1)]);
      compute_local[(2)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(2)], compute_local[(2)]);
      compute_local[(3)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(0)], compute_local[(3)]);
      compute_local[(4)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(1)], compute_local[(4)]);
      compute_local[(5)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(2)], compute_local[(5)]);
      compute_local[(6)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(0)], compute_local[(6)]);
      compute_local[(7)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(1)], compute_local[(7)]);
      compute_local[(8)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(2)], compute_local[(8)]);
      compute_local[(9)] = __ocml_fma_f32(placeholder_shared_local[(3)], placeholder_d_shared_local[(0)], compute_local[(9)]);
      compute_local[(10)] = __ocml_fma_f32(placeholder_shared_local[(3)], placeholder_d_shared_local[(1)], compute_local[(10)]);
      compute_local[(11)] = __ocml_fma_f32(placeholder_shared_local[(3)], placeholder_d_shared_local[(2)], compute_local[(11)]);
      compute_local[(12)] = __ocml_fma_f32(placeholder_shared_local[(4)], placeholder_d_shared_local[(0)], compute_local[(12)]);
      compute_local[(13)] = __ocml_fma_f32(placeholder_shared_local[(4)], placeholder_d_shared_local[(1)], compute_local[(13)]);
      compute_local[(14)] = __ocml_fma_f32(placeholder_shared_local[(4)], placeholder_d_shared_local[(2)], compute_local[(14)]);
    }
  }
  compute[((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)))] = compute_local[(0)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1))] = compute_local[(1)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 2))] = compute_local[(2)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 480))] = compute_local[(3)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 481))] = compute_local[(4)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 482))] = compute_local[(5)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 960))] = compute_local[(6)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 961))] = compute_local[(7)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 962))] = compute_local[(8)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1440))] = compute_local[(9)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1441))] = compute_local[(10)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1442))] = compute_local[(11)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1920))] = compute_local[(12)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1921))] = compute_local[(13)];
  compute[(((((((((int)task_idx.z) * 230400) + (((int)task_idx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)task_idx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1922))] = compute_local[(14)];
}

__device__  void fused_nn_softmax_1_kernel1_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_softmax_exp, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem){
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 43; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    if ((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_softmax_exp[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = __ocml_exp_f32((placeholder[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] - T_softmax_maxelem[(((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480))]));
    }
  }
}

__device__  void fused_reshape_add_multiply_erf_multiply_add_multiply_reshape_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_reshape, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = ((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480))]) * __ocml_fma_f32(__ocml_erf_f32(((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480))]) * 7.071068e-01f)), 5.000000e-01f, 5.000000e-01f));
    }
  }
}

__device__  void fused_mean_1_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ placeholder, float* __restrict__ placeholder_red){
  float placeholder_red_rf[1];
  __shared__ float red_buf0[1024];
  placeholder_red_rf[(0)] = 0.000000e+00f;
  for (int k2_outer = 0; k2_outer < 15; ++k2_outer) {
    placeholder_red_rf[(0)] = (placeholder_red_rf[(0)] + placeholder[(((((((int)task_idx.x) * 15360) + (((int)threadIdx.y) * 480)) + (k2_outer * 32)) + ((int)threadIdx.x)))]);
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = placeholder_red_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 16))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 8))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 4))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 2))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    placeholder_red[(((((int)task_idx.x) * 32) + ((int)threadIdx.y)))] = ((volatile float*)red_buf0)[((((int)threadIdx.y) * 32))];
  }
}

__device__  void fused_subtract_add_sqrt_divide_multiply_add_1_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_add[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = __ocml_fma_f32(((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] - placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480))]) / __ocml_sqrt_f32((placeholder2[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480))] + 1.000000e-12f))), placeholder3[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 480))], placeholder4[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 480))]);
    }
  }
}

__device__  void fused_nn_batch_matmul_3_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute){
  float compute_local[9];
  __shared__ float placeholder_shared[2400];
  __shared__ float placeholder_d_shared[300];
  float placeholder_shared_local[3];
  float placeholder_d_shared_local[3];
  for (int i_c_init = 0; i_c_init < 3; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 3; ++j_c_init) {
      compute_local[(((i_c_init * 3) + j_c_init))] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 24; ++k_outer) {
    __syncthreads();
    placeholder_shared[(((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 1))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 2))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 3))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 20))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 480))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 21))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 481))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 22))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 482))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 23))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 483))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 40))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 960))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 41))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 961))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 42))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 962))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 43))] = placeholder[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 963))];
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))] = placeholder1[(((((((int)task_idx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)))];
    }
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 1))] = placeholder1[((((((((int)task_idx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 1))];
    }
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 2))] = placeholder1[((((((((int)task_idx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 2))];
    }
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 3))] = placeholder1[((((((((int)task_idx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 3))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 20; ++k_inner) {
      placeholder_shared_local[(0)] = placeholder_shared[(((((int)threadIdx.y) * 60) + k_inner))];
      placeholder_shared_local[(1)] = placeholder_shared[((((((int)threadIdx.y) * 60) + k_inner) + 20))];
      placeholder_shared_local[(2)] = placeholder_shared[((((((int)threadIdx.y) * 60) + k_inner) + 40))];
      placeholder_d_shared_local[(0)] = placeholder_d_shared[(((((int)threadIdx.x) * 60) + k_inner))];
      placeholder_d_shared_local[(1)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 20))];
      placeholder_d_shared_local[(2)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 40))];
      compute_local[(0)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(0)], compute_local[(0)]);
      compute_local[(1)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(1)], compute_local[(1)]);
      compute_local[(2)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(2)], compute_local[(2)]);
      compute_local[(3)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(0)], compute_local[(3)]);
      compute_local[(4)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(1)], compute_local[(4)]);
      compute_local[(5)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(2)], compute_local[(5)]);
      compute_local[(6)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(0)], compute_local[(6)]);
      compute_local[(7)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(1)], compute_local[(7)]);
      compute_local[(8)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(2)], compute_local[(8)]);
    }
  }
  compute[(((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)task_idx.x) * 15)) + (((int)threadIdx.x) * 3)))] = compute_local[(0)];
  compute[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)task_idx.x) * 15)) + (((int)threadIdx.x) * 3)) + 1))] = compute_local[(1)];
  compute[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)task_idx.x) * 15)) + (((int)threadIdx.x) * 3)) + 2))] = compute_local[(2)];
  compute[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)task_idx.x) * 15)) + (((int)threadIdx.x) * 3)) + 480))] = compute_local[(3)];
  compute[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)task_idx.x) * 15)) + (((int)threadIdx.x) * 3)) + 481))] = compute_local[(4)];
  compute[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)task_idx.x) * 15)) + (((int)threadIdx.x) * 3)) + 482))] = compute_local[(5)];
  compute[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)task_idx.x) * 15)) + (((int)threadIdx.x) * 3)) + 960))] = compute_local[(6)];
  compute[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)task_idx.x) * 15)) + (((int)threadIdx.x) * 3)) + 961))] = compute_local[(7)];
  compute[((((((((int)task_idx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)task_idx.x) * 15)) + (((int)threadIdx.x) * 3)) + 962))] = compute_local[(8)];
}

__device__  void fused_reshape_5_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_reshape, float* __restrict__ placeholder){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 43; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))];
    }
  }
}

__device__  void fused_reshape_transpose_copy_reshape_1_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_reshape, float* __restrict__ placeholder){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 480) / 40) * 19200) + (((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480) * 40)) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 40)))];
    }
  }
}

__device__  void fused_reshape_add_reshape_transpose_reshape_transpose_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_transpose, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_transpose[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = (placeholder[(((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 480) * 480) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 480))]);
    }
  }
}

__device__  void fused_reshape_add_reshape_transpose_transpose_reshape_transpose_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_transpose, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_transpose[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = (placeholder[(((((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40) * 480) + (((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 19200) * 40)) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 40)))] + placeholder1[((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40))]);
    }
  }
}

__device__  void fused_reshape_add_reshape_transpose_divide_reshape_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_reshape, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = ((placeholder[(((((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40) * 480) + (((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 19200) * 40)) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 40)))] + placeholder1[((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40))]) * 1.581139e-01f);
    }
  }
}

__device__  void fused_full_equal_reshape_kernel0_device(dim3 task_idx, dim3 thread_idx, signed char* __restrict__ T_reshape){
  if (((((int)task_idx.x) * 256) + ((int)threadIdx.x)) < 480) {
    T_reshape[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))] = (signed char)0;
  }
}

__device__  void fused_cast_take_broadcast_to_like_cast_take_add_1_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_add, float* __restrict__ placeholder, long* __restrict__ placeholder1, float* __restrict__ placeholder2, long* __restrict__ placeholder3){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_add[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = (placeholder[(((min(max(0, ((int)placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480))])), 30521) * 480) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 480)))] + placeholder2[(((min(max(0, ((int)placeholder3[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480))])), 1023) * 480) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 480)))]);
    }
  }
}

__device__  void fused_nn_softmax_1_kernel2_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_exp){
  if (((((int)task_idx.x) * 256) + ((int)threadIdx.x)) < 5760) {
    T_softmax_maxelem[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))] = 0.000000e+00f;
  }
  for (int k = 0; k < 480; ++k) {
    if (((((int)task_idx.x) * 256) + ((int)threadIdx.x)) < 5760) {
      T_softmax_maxelem[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))] = (T_softmax_maxelem[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))] + T_softmax_exp[((((((int)task_idx.x) * 122880) + (((int)threadIdx.x) * 480)) + k))]);
    }
  }
}

__device__  void fused_mean_1_kernel1_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_divide, float* __restrict__ placeholder_red){
  if (((((int)task_idx.x) * 256) + ((int)threadIdx.x)) < 480) {
    T_divide[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))] = (placeholder_red[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))] * 2.083333e-03f);
  }
}

__device__  void fused_reshape_4_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_reshape, float* __restrict__ placeholder){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))];
    }
  }
}

__device__  void fused_variance_1_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_multiply_red){
  float T_multiply_red_rf[1];
  __shared__ float red_buf0[1024];
  T_multiply_red_rf[(0)] = 0.000000e+00f;
  for (int k2_outer = 0; k2_outer < 15; ++k2_outer) {
    T_multiply_red_rf[(0)] = __ocml_fma_f32((placeholder[(((((((int)task_idx.x) * 15360) + (((int)threadIdx.y) * 480)) + (k2_outer * 32)) + ((int)threadIdx.x)))] - placeholder1[(((((int)task_idx.x) * 32) + ((int)threadIdx.y)))]), (placeholder[(((((((int)task_idx.x) * 15360) + (((int)threadIdx.y) * 480)) + (k2_outer * 32)) + ((int)threadIdx.x)))] - placeholder1[(((((int)task_idx.x) * 32) + ((int)threadIdx.y)))]), T_multiply_red_rf[(0)]);
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = T_multiply_red_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 16))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 8))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 4))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 2))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_multiply_red[(((((int)task_idx.x) * 32) + ((int)threadIdx.y)))] = ((volatile float*)red_buf0)[((((int)threadIdx.y) * 32))];
  }
}

__device__  void fused_variance_1_kernel1_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_divide, float* __restrict__ T_multiply_red){
  if (((((int)task_idx.x) * 256) + ((int)threadIdx.x)) < 480) {
    T_divide[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))] = (T_multiply_red[(((((int)task_idx.x) * 256) + ((int)threadIdx.x)))] * 2.083333e-03f);
  }
}

__device__  void fused_reshape_cast_broadcast_to_like_where_kernel0_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_where, signed char* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 43; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_where[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = ((((int)((bool)placeholder[(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) % 480))])) != 0) ? -__int_as_float(0x7f800000) : placeholder1[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))]);
    }
  }
}

__device__  void fused_nn_softmax_1_kernel3_device(dim3 task_idx, dim3 thread_idx, float* __restrict__ T_softmax_norm, float* __restrict__ T_softmax_exp, float* __restrict__ T_softmax_maxelem){
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 43; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    if ((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_softmax_norm[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] = (T_softmax_exp[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)))] / T_softmax_maxelem[(((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)task_idx.x) * 256)) + ((int)threadIdx.x)) / 480))]);
    }
  }
}


extern "C" __global__ void fused_nn_softmax_1_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(23,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_nn_softmax_1_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1]);
    }
}

extern "C" __global__ void fused_reshape_add_add_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_reshape_add_add_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2], (float* __restrict__)args[3]);
    }
}

extern "C" __global__ void fused_nn_batch_matmul_4_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(2,10,12);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_nn_batch_matmul_4_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void fused_nn_batch_matmul_5_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(10,6,12);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_nn_batch_matmul_5_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void fused_nn_softmax_1_kernel1(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_nn_softmax_1_kernel1_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void fused_reshape_add_multiply_erf_multiply_add_multiply_reshape_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_reshape_add_multiply_erf_multiply_add_multiply_reshape_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void fused_mean_1_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(15,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_mean_1_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1]);
    }
}

extern "C" __global__ void fused_subtract_add_sqrt_divide_multiply_add_1_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_subtract_add_sqrt_divide_multiply_add_1_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2], (float* __restrict__)args[3], (float* __restrict__)args[4], (float* __restrict__)args[5]);
    }
}

extern "C" __global__ void fused_nn_batch_matmul_3_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(32,4,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_nn_batch_matmul_3_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void fused_reshape_5_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_reshape_5_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1]);
    }
}

extern "C" __global__ void fused_reshape_transpose_copy_reshape_1_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_reshape_transpose_copy_reshape_1_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1]);
    }
}

extern "C" __global__ void fused_reshape_add_reshape_transpose_reshape_transpose_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_reshape_add_reshape_transpose_reshape_transpose_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void fused_reshape_add_reshape_transpose_transpose_reshape_transpose_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_reshape_add_reshape_transpose_transpose_reshape_transpose_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void fused_reshape_add_reshape_transpose_divide_reshape_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_reshape_add_reshape_transpose_divide_reshape_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void fused_full_equal_reshape_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(2,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_full_equal_reshape_kernel0_device(task_idx, thread_idx, (signed char* __restrict__)args[0]);
    }
}

extern "C" __global__ void fused_cast_take_broadcast_to_like_cast_take_add_1_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_cast_take_broadcast_to_like_cast_take_add_1_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (long* __restrict__)args[2], (float* __restrict__)args[3], (long* __restrict__)args[4]);
    }
}

extern "C" __global__ void fused_nn_softmax_1_kernel2(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(23,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_nn_softmax_1_kernel2_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1]);
    }
}

extern "C" __global__ void fused_mean_1_kernel1(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(2,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_mean_1_kernel1_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1]);
    }
}

extern "C" __global__ void fused_reshape_4_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_reshape_4_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1]);
    }
}

extern "C" __global__ void fused_variance_1_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(15,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_variance_1_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void fused_variance_1_kernel1(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(2,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_variance_1_kernel1_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1]);
    }
}

extern "C" __global__ void fused_reshape_cast_broadcast_to_like_where_kernel0(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_reshape_cast_broadcast_to_like_where_kernel0_device(task_idx, thread_idx, (float* __restrict__)args[0], (signed char* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void fused_nn_softmax_1_kernel3(volatile int* preempted, int task_num, int task_offset, int* task_slot, float** args) {
    dim3 task_dim = dim3(256,1,1);
    dim3 thread_idx = dim3(threadIdx.x, threadIdx.y, threadIdx.z);
    __shared__ int idx[1];
    while (true) {
        if (*preempted) return;
        if (threadIdx.x + threadIdx.y + threadIdx.z == 0) {
            // first thread of the block
            int temp = atomicAdd(task_slot, 1);
            idx[0] = temp + task_offset;
        }
        __syncthreads();
        if (idx[0] >= task_num) return;
        dim3 task_idx = get_3d_idx(idx[0], task_dim);
        fused_nn_softmax_1_kernel3_device(task_idx, thread_idx, (float* __restrict__)args[0], (float* __restrict__)args[1], (float* __restrict__)args[2]);
    }
}

extern "C" __global__ void preemption_proxy(volatile int* stop, volatile int* host, volatile int* device) {
    while(!(*stop)) {
        *device = *host;
        long long start_clock = clock64();
        while (clock64() < (start_clock+10000)) {}
    }
}
