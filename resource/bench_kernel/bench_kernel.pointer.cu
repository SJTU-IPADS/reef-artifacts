#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))


#define NUM_BLOCKS 64
#define NUM_TREHAD_PER_BLOCK 128
#define BLOCKDIM_X 4
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 4

#define __activemask() 0

#define CU_NUM 60

__device__ __forceinline__ bool is_first_thread() {
  return threadIdx.x == 0;
}

__device__ __forceinline__ unsigned int get_cu_id() {
  return blockIdx.x % CU_NUM;
}

__device__ __forceinline__ dim3 get_3d_idx(int idx, dim3 dim) {
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__device__ __noinline__ void loop_device(float* __restrict__ n, float* __restrict__ b, float* __restrict__ temp){
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    for (int i = 0; i < n[0]; i++)
        temp[i % 1024] = i / (i + 1) * b[arrayOffset];
}


extern "C" __global__ __attribute__((amdgpu_num_vgpr(24))) __attribute__((amdgpu_num_sgpr(30)))void loop(float* __restrict__ n, float* __restrict__ b, float* __restrict__ temp) {
    loop_device(n, b, temp);
}

extern "C" __global__ void _get_func_pointers(unsigned long long int *address) {
    address[0] = (unsigned long long int)&loop_device;

}

