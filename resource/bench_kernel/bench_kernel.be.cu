#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))


#define NUM_BLOCKS 64
#define NUM_TREHAD_PER_BLOCK 128
#define BLOCKDIM_X 4
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 4

#define __activemask() 0

__device__ void loop_device(float* __restrict__ n, float* __restrict__ b, float* __restrict__ temp){
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    for (int i = 0; i < n[0]; i++)
        temp[i % 1024] = i / (i + 1) * b[arrayOffset];
}


extern "C" __global__ void loop(int* preempted, int* task_slot, float* __restrict__ n, float* __restrict__ b, float* __restrict__ temp) {
    if (*preempted) return;
    loop_device(n, b, temp);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        
