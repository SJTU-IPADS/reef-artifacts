#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))


#define NUM_BLOCKS 64
#define NUM_TREHAD_PER_BLOCK 128
#define BLOCKDIM_X 4
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 4

#define __activemask() 0

extern "C" __global__ void loop(float* __restrict__ n, float* __restrict__ b, float* __restrict__ temp) {
    if((int)threadIdx.x + (int)threadIdx.y + (int)threadIdx.z + (int)blockIdx.x + (int)blockIdx.y + (int)blockIdx.z < 0) return;
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    for (int i = 0; i < n[0]; i++)
        temp[i % 1024] = i / (i + 1) * b[arrayOffset];
}


#define CALL_FRAMEWORK(idx) \
extern "C" __global__ void proxy_kernel_##idx(\
  void* func, float** param) \
{\
  asm volatile(\
    "  s_load_dwordx4 s[16:19], s[4:5], 0x0\n"\
    "  s_waitcnt lgkmcnt(0)\n"\
    "  s_mov_b64 s[4:5], s[18:19]\n"\
    "  s_setpc_b64 s[16:17]\n"\
    "  s_endpgm\n"\
  );\
}    

CALL_FRAMEWORK(1)
CALL_FRAMEWORK(2)
CALL_FRAMEWORK(3)
CALL_FRAMEWORK(4)
CALL_FRAMEWORK(5)
CALL_FRAMEWORK(6)
CALL_FRAMEWORK(7)
CALL_FRAMEWORK(8)
CALL_FRAMEWORK(9)
CALL_FRAMEWORK(10)
CALL_FRAMEWORK(nostack_1)
CALL_FRAMEWORK(nostack_2)
CALL_FRAMEWORK(nostack_3)
CALL_FRAMEWORK(nostack_4)
CALL_FRAMEWORK(nostack_5)
CALL_FRAMEWORK(nostack_6)
CALL_FRAMEWORK(nostack_7)
CALL_FRAMEWORK(nostack_8)
CALL_FRAMEWORK(nostack_9)
CALL_FRAMEWORK(nostack_10)
