#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))


#define NUM_BLOCKS 64
#define NUM_TREHAD_PER_BLOCK 128
#define BLOCKDIM_X 4
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 4

#define __activemask() 0

extern "C" __global__ void loop(float* __restrict__ n, float* __restrict__ b, float* __restrict__ temp) {
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    for (int i = 0; i < n[0]; i++)
        temp[i % 1024] = i / (i + 1) * b[arrayOffset];
}

